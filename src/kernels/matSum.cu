
#include <hip/hip_runtime.h>
extern "C" __global__ void matSum(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}