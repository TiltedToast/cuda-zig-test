
#include <hip/hip_runtime.h>
extern "C" __global__ void VecAdd(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}